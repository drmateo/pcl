#include "hip/hip_runtime.h"
/*
 * Software License Agreement (BSD License)
 *
 *  Point Cloud Library (PCL) - www.pointclouds.org
 *  Copyright (c) 2011, Willow Garage, Inc.
 * 
 *  All rights reserved.
 *
 *  Redistribution and use in source and binary forms, with or without
 *  modification, are permitted provided that the following conditions
 *  are met:
 *
 *   * Redistributions of source code must retain the above copyright
 *     notice, this list of conditions and the following disclaimer.
 *   * Redistributions in binary form must reproduce the above
 *     copyright notice, this list of conditions and the following
 *     disclaimer in the documentation and/or other materials provided
 *     with the distribution.
 *   * Neither the name of Willow Garage, Inc. nor the names of its
 *     contributors may be used to endorse or promote products derived
 *     from this software without specific prior written permission.
 *
 *  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
 *  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
 *  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS
 *  FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE
 *  COPYRIGHT OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT,
 *  INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
 *  BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 *  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 *  CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 *  LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN
 *  ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 *  POSSIBILITY OF SUCH DAMAGE.
 *
 */

#include "device.hpp"

using namespace pcl::device;

namespace pcl
{
namespace device
{
struct ImageGenerator
{
	enum
	{
		CTA_SIZE_X = 32, CTA_SIZE_Y = 8
	};

	PtrStep<float> vmap;
	PtrStep<float> nmap;

	LightSource light;

	mutable PtrStepSz<uchar3> dst;

	__device__ __forceinline__ void
	operator () () const
	{
		int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
		int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;

		if (x >= dst.cols || y >= dst.rows)
			return;

		float3 v, n;
		v.x = vmap.ptr (y)[x];
		n.x = nmap.ptr (y)[x];

		uchar3 color = make_uchar3 (0, 0, 0);

		if (!isnan (v.x) && !isnan (n.x))
		{
			v.y = vmap.ptr (y + dst.rows)[x];
			v.z = vmap.ptr (y + 2 * dst.rows)[x];

			n.y = nmap.ptr (y + dst.rows)[x];
			n.z = nmap.ptr (y + 2 * dst.rows)[x];

			float weight = 1.f;

			for (int i = 0; i < light.number; ++i)
			{
				float3 vec = normalized (light.pos[i] - v);

				weight *= fabs (dot (vec, n));
			}

			int br = (int)(205 * weight) + 50;
			br = max (0, min (255, br));
			color = make_uchar3 (br, br, br);
		}
		dst.ptr (y)[x] = color;
	}
};

__global__ void
generateImageKernel (const ImageGenerator ig) {
	ig ();
}
}
}


void
pcl::device::generateImage (const MapArr& vmap, const MapArr& nmap, const LightSource& light, 
		PtrStepSz<uchar3> dst)
{
	ImageGenerator ig;
	ig.vmap = vmap;
	ig.nmap = nmap;
	ig.light = light;
	ig.dst = dst;

	dim3 block (ImageGenerator::CTA_SIZE_X, ImageGenerator::CTA_SIZE_Y);
	dim3 grid (divUp (dst.cols, block.x), divUp (dst.rows, block.y));

	generateImageKernel<<<grid, block>>>(ig);
	cudaSafeCall (hipGetLastError ());
	cudaSafeCall (hipDeviceSynchronize ());
} 

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////

namespace pcl
{
namespace device
{
__global__ void generateDepthKernel(const float3 R_inv_row3, const float3 t, const PtrStep<float> vmap, PtrStepSz<unsigned short> depth)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < depth.cols && y < depth.rows)
	{
		unsigned short result = 0;

		float3 v_g;
		v_g.x = vmap.ptr (y)[x];
		if (!isnan (v_g.x))
		{
			v_g.y = vmap.ptr (y +     depth.rows)[x];
			v_g.z = vmap.ptr (y + 2 * depth.rows)[x];

			float v_z = dot(R_inv_row3, v_g - t);

			result = static_cast<unsigned short>(v_z * 1000);
		}
		depth.ptr(y)[x] = result;
	}
}
}
}

void
pcl::device::generateDepth (const Mat33& R_inv, const float3& t, const MapArr& vmap, DepthMap& dst)
{
	dim3 block(32, 8);
	dim3 grid(divUp(dst.cols(), block.x), divUp(dst.rows(), block.y));

	generateDepthKernel<<<grid, block>>>(R_inv.data[2], t, vmap, dst);
	cudaSafeCall (hipGetLastError ());
	cudaSafeCall (hipDeviceSynchronize ());
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////

namespace pcl
{
namespace device
{
struct GenerateDepth
{
	Mat33 R_inv;
	float3 R_inv_row3;

	float3 t;
	PtrStep<float> vmap;
	mutable PtrStepSz<unsigned short> depth;



	Intr intr;

	Mat33 RBase;
	float3 tBase;



	float z_min ;
	float z_max ;

	__device__ __forceinline__ void
	operator () () const
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if (x < depth.cols && y < depth.rows)
		{
			unsigned short result = 0;
			float v_z;
			float3 v_g;
			v_g.x = vmap.ptr (y)[x];
			if (!isnan (v_g.x))
			{
				v_g.y = vmap.ptr (y +     depth.rows)[x];
				v_g.z = vmap.ptr (y + 2 * depth.rows)[x];

				v_z = dot(R_inv_row3, v_g - t);

				result = static_cast<unsigned short>(v_z * 1000);
			}

			if (result != 0)
			{
				float3 pt ;

				pt.z = v_z * 1000.f;
				pt.x = ((float)x - intr.cx) * pt.z / intr.fx;
				pt.y = ((float)y - intr.cy) * pt.z / intr.fy;

				pt *= 0.001f;

				//            				pt.x = v_g.x ;
				//            				pt.y = v_g.y ;
				//            				pt.z = v_z ;

				//            				float3 pt_t = R_inv * (pt - t);
				float3 pt_t = RBase * (pt - tBase);


				if (pt_t.z > z_max || pt_t.z < z_min)
					result = 0;
			}

			depth.ptr(y)[x] = result;
		}
	}       /* operator() */

};

struct GenerateDepthV2 : GenerateDepth
{
	float3 pt1;
	float3 pt2;
	float lengthsq;
	float radius_sq;


	__device__ __forceinline__ float
	CylTest( float3 testpt ) const
	{
		float dx, dy, dz;	// vector d  from line segment point 1 to point 2
		float pdx, pdy, pdz;	// vector pd from point 1 to test point
		float dot, dsq;

		dx = pt2.x - pt1.x;	// translate so pt1 is origin.  Make vector from
		dy = pt2.y - pt1.y;     // pt1 to pt2.  Need for this is easily eliminated
		dz = pt2.z - pt1.z;

		pdx = testpt.x - pt1.x;		// vector from pt1 to test point.
		pdy = testpt.y - pt1.y;
		pdz = testpt.z - pt1.z;

		dot = pdx * dx + pdy * dy + pdz * dz;

		if( dot < 0.0f || dot > lengthsq )
		{
			return( -1.0f );
		}
		else
		{
			dsq = (pdx*pdx + pdy*pdy + pdz*pdz) - dot*dot/lengthsq;

			if( dsq > radius_sq )
			{
				return( -1.0f );
			}
			else
			{
				return( dsq );		// return distance squared to axis
			}
		}
	}

	__device__ __forceinline__ void
	operator () () const
	{
		int x = threadIdx.x + blockIdx.x * blockDim.x;
		int y = threadIdx.y + blockIdx.y * blockDim.y;

		if (x < depth.cols && y < depth.rows)
		{
			unsigned short result = 0;
			float v_z;
			float3 v_g;
			v_g.x = vmap.ptr (y)[x];
			if (!isnan (v_g.x))
			{
				v_g.y = vmap.ptr (y +     depth.rows)[x];
				v_g.z = vmap.ptr (y + 2 * depth.rows)[x];

				v_z = dot(R_inv_row3, v_g - t);

				result = static_cast<unsigned short>(v_z * 1000);
			}

			if (result != 0)
			{
				float3 pt ;

				pt.z = v_z * 1000.f;
				pt.x = ((float)x - intr.cx) * pt.z / intr.fx;
				pt.y = ((float)y - intr.cy) * pt.z / intr.fy;

				pt *= 0.001f;

				//            				pt.x = v_g.x ;
				//            				pt.y = v_g.y ;
				//            				pt.z = v_z ;

				//            				float3 pt_t = R_inv * (pt - t);
				float3 pt_t = RBase * pt + tBase;


				if (CylTest( pt_t ) != -1.f || pt_t.z > z_max || pt_t.z < z_min)
					result = 0;
			}

			depth.ptr(y)[x] = result;
		}
	}       /* operator() */
};

__global__ void
generateDepthKernel (const GenerateDepth gd) {
	gd ();
}

__global__ void
generateDepthKernel (const GenerateDepthV2 gd) {
	gd ();
}
}
}

void
pcl::device::generateDepth (const Intr& intr, const Mat33& R_inv, const float3& t, const Mat33& R_base, const float3& t_base, float z_min, float z_max, const MapArr& vmap, DepthMap& dst)
{
	dim3 block(32, 8);
	dim3 grid(divUp(dst.cols(), block.x), divUp(dst.rows(), block.y));

	GenerateDepth gd;
	gd.R_inv = R_inv;
	gd.R_inv_row3 = R_inv.data[2];
	gd.t = t;
	gd.RBase = R_base;
	gd.tBase = t_base;
	gd.vmap = vmap;
	gd.depth = dst;
	gd.intr = intr;
	gd.z_min = z_min;
	gd.z_max = z_max;

	generateDepthKernel<<<grid, block>>>(gd);
	cudaSafeCall (hipGetLastError ());
	cudaSafeCall (hipDeviceSynchronize ());
}

void
pcl::device::generateDepth (const Intr& intr, const Mat33& R_inv, const float3& t, const Mat33& R_base, const float3& t_base, float z_min, float z_max,
		   const float3& pt1, const float3& pt2, float lengthsq, float radius_sq,  const MapArr& vmap, DepthMap& dst)
{
	dim3 block(32, 8);
	dim3 grid(divUp(dst.cols(), block.x), divUp(dst.rows(), block.y));

	GenerateDepthV2 gd;
	gd.R_inv = R_inv;
	gd.R_inv_row3 = R_inv.data[2];
	gd.t = t;
	gd.RBase = R_base;
	gd.tBase = t_base;
	gd.vmap = vmap;
	gd.depth = dst;
	gd.intr = intr;
	gd.z_min = z_min;
	gd.z_max = z_max;


	gd.pt1 = pt1;
	gd.pt2 = pt2;
	gd.lengthsq = lengthsq;
	gd.radius_sq = radius_sq;

	generateDepthKernel<<<grid, block>>>(gd);
	cudaSafeCall (hipGetLastError ());
	cudaSafeCall (hipDeviceSynchronize ());
}

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////

namespace pcl
{
namespace device
{
__global__ void
paint3DViewKernel(const PtrStep<uchar3> colors, PtrStepSz<uchar3> dst, float colors_weight)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < dst.cols && y < dst.rows)
	{
		uchar3 value = dst.ptr(y)[x];
		uchar3 color = colors.ptr(y)[x];

		if (value.x != 0 || value.y != 0 || value.z != 0)
		{
			float cx = value.x * (1.f - colors_weight) + color.x * colors_weight;
			float cy = value.y * (1.f - colors_weight) + color.y * colors_weight;
			float cz = value.z * (1.f - colors_weight) + color.z * colors_weight;

			value.x = min(255, max(0, __float2int_rn(cx)));
			value.y = min(255, max(0, __float2int_rn(cy)));
			value.z = min(255, max(0, __float2int_rn(cz)));
		}

		dst.ptr(y)[x] = value;
	}
}
}
}

void 
pcl::device::paint3DView(const PtrStep<uchar3>& colors, PtrStepSz<uchar3> dst, float colors_weight)
{
	dim3 block(32, 8);
	dim3 grid(divUp(dst.cols, block.x), divUp(dst.rows, block.y));

	colors_weight = min(1.f, max(0.f, colors_weight));

	paint3DViewKernel<<<grid, block>>>(colors, dst, colors_weight);
	cudaSafeCall (hipGetLastError ());
	cudaSafeCall (hipDeviceSynchronize ());
}
